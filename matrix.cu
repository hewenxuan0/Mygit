#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
 
 
__global__ void MatMul(int *M,int *N,int *P,int width)
{
	int x = threadIdx.x;
	int y = threadIdx.y;
	
	
	float elem1 = 0.0,elem2 = 0.0,value = 0.0;
	for(int i = 0;i < width;i++)
	{
		elem1 = M[y * width + i];//取M矩阵的一行
		elem2 = N[i * width + x];//取N矩阵的一列
		
		value += elem1 * elem2;//求和
	}
	
	P[y * width + x] = value;
}

void useCUDA(int *a,int *b,int *c,int width)
{
    printf("width=%d",width);
    for(int i=0;i<width*width;i++)
        printf("a=%d",a[i]);
    
    //const int width = 30;
	//int a[width][width],b[width][width],c[width][width];
	int *M,*N,*P;
	
	//int width = width;
	//int NUM = 900;
	dim3 blockSize(width,width);
	
	hipEvent_t start,stop;
	float elapsedTime = 0;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	
	//设备端内存分配
	hipMalloc((void**)&M,width * width * sizeof(int));
	hipMalloc((void**)&N,width * width * sizeof(int));
	hipMalloc((void**)&P,width * width * sizeof(int));
	
	//初始化
	
	
	int Size = width * width;
	//数据拷贝，主机到设备
	hipMemcpy(M,a,Size * sizeof(int),hipMemcpyHostToDevice);
	hipMemcpy(N,b,Size * sizeof(int),hipMemcpyHostToDevice);
	
	hipEventRecord(start,0);
	MatMul<<<1,blockSize>>>(M,N,P,width);//调用核函数
	hipDeviceSynchronize();
	hipEventRecord(stop,0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&elapsedTime,start,stop);
	
	hipMemcpy(c,P,Size * sizeof(int),hipMemcpyDeviceToHost);
	
	for(int i=0;i<width;i++){
		for(int j=0;j<width;j++){
			//printf("c=%d \n",c[0][0]);
		}
		printf("\n");
	}
	//printf("time=%d \n",elapsedTime);
	
	
	//释放设备内存
	hipFree(M);
	hipFree(N);
	hipFree(P);

}
 
int main()
{
	const int ND = 30;
	//int a[ND][ND],b[ND][ND],c[ND][ND];
	int *a=new int[ND*ND],*b=new int[ND*ND],*c=new int[ND*ND];

	for(int i = 0;i < ND;i++)
	{
		for(int j = 0;j < ND;j++)
		{
			// a[i][j] = 2;
			// b[i][j] = 3;
			a[i*ND+j]=2;
			b[i*ND+j]=3;
		}
	}

	useCUDA(a,b,c,ND);
	// int *M,*N,*P;
	
	// int width = ND;
	// //int NUM = 900;
	// dim3 blockSize(ND,ND);
	
	// cudaEvent_t start,stop;
	// float elapsedTime = 0;
	// cudaEventCreate(&start);
	// cudaEventCreate(&stop);
	
	// //设备端内存分配
	// cudaMalloc((void**)&M,ND * ND * sizeof(int));
	// cudaMalloc((void**)&N,ND * ND * sizeof(int));
	// cudaMalloc((void**)&P,ND * ND * sizeof(int));
	
	// //初始化
	
	
	// int Size = ND * ND;
	// //数据拷贝，主机到设备
	// cudaMemcpy(M,a,Size * sizeof(int),cudaMemcpyHostToDevice);
	// cudaMemcpy(N,b,Size * sizeof(int),cudaMemcpyHostToDevice);
	
	// cudaEventRecord(start,0);
	// MatMul<<<1,blockSize>>>(M,N,P,width);//调用核函数
	// cudaDeviceSynchronize();
	// cudaEventRecord(stop,0);
	// cudaEventSynchronize(stop);
	// cudaEventElapsedTime(&elapsedTime,start,stop);
	
	// cudaMemcpy(c,P,Size * sizeof(int),cudaMemcpyDeviceToHost);
	
	// // for(int i=0;i<ND;i++){
	// // 	for(int j=0;j<ND;j++){
	// // 		//printf("c=%d \n",c[0][0]);
	// // 	}
	// // 	printf("hello\n");
	// // }
	// printf("time=%f \n",elapsedTime);
	
	
	// //释放设备内存
	// cudaFree(M);
	// cudaFree(N);
	// cudaFree(P);
	
	// return 0;
}