#include "hip/hip_runtime.h"
#include "foo.cuh"


#define CHECK(res) { if(res != hipSuccess){printf("Error :%s:%d , ", __FILE__,__LINE__);   \
printf("code : %d , reason : %s \n", res,hipGetErrorString(res));exit(-1);}}


__global__ void MatMul(int *M,int *N,int *P,int width)
{
	int x = threadIdx.x;
	int y = threadIdx.y;
	
	
	float elem1 = 0.0,elem2 = 0.0,value = 0.0;
	for(int i = 0;i < width;i++)
	{
		elem1 = M[y * width + i];//取M矩阵的一行
		elem2 = N[i * width + x];//取N矩阵的一列
		
		value += elem1 * elem2;//求和
	}
	
	P[y * width + x] = value;
}


void useCUDA(int *a,int *b,int *c,int width)
{
    // printf("width=%d",width);
    // for(int i=0;i<width*width;i++)
    //     printf("a=%d",a[i]);
    
    //const int width = 30;
	//int a[width][width],b[width][width],c[width][width];
	int *M,*N,*P;
	
	//int width = width;
	//int NUM = 900;
	dim3 blockSize(width,width);
	
	hipEvent_t start,stop;
	float elapsedTime = 0;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	
	//设备端内存分配
	hipMalloc((void**)&M,width * width * sizeof(int));
	hipMalloc((void**)&N,width * width * sizeof(int));
	hipMalloc((void**)&P,width * width * sizeof(int));
	
	//初始化
	
	
	int Size = width * width;
	//数据拷贝，主机到设备
	hipMemcpy(M,a,Size * sizeof(int),hipMemcpyHostToDevice);
	hipMemcpy(N,b,Size * sizeof(int),hipMemcpyHostToDevice);
	
	hipEventRecord(start,0);
	MatMul<<<1,blockSize>>>(M,N,P,width);//调用核函数
	hipDeviceSynchronize();
	hipEventRecord(stop,0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&elapsedTime,start,stop);
	
	hipMemcpy(c,P,Size * sizeof(int),hipMemcpyDeviceToHost);
	
	for(int i=0;i<width;i++){
		for(int j=0;j<width;j++){
			printf("c=%d \n",c[i*width+j]);
		}
		printf("\n");
	}
	//printf("time=%d \n",elapsedTime);
	
	
	//释放设备内存
	hipFree(M);
	hipFree(N);
	hipFree(P);

}
